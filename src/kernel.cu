#include "hip/hip_runtime.h"
//
// Created by hieu on 9/16/25.
//

// #include<iostream>
#include"kernel.cuh"

#include <cmath>
#include <format>
#include<iostream>
__global__ void hello_kernel() {
	printf("Hello World from GPU!\n");
}

__global__ auto vec_add_kernel(
	const float *a,
	const float *b,
	float *c,
	const size_t n
) -> void {
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n)
		c[i] = a[i] + b[i];
}

auto vec_add_cuda(
	const float *a,
	const float *b,
	float *c,
	const size_t n,
	size_t block_size
) -> void {
	float *a_d{nullptr}, *b_d{nullptr}, *c_d{nullptr};
	size_t size = sizeof(float) * n;

	auto malloc_device = [size](float **v)-> void {
		hipMalloc(reinterpret_cast<void **>(v), size);
	};
	malloc_device(&a_d);
	malloc_device(&b_d);
	malloc_device(&c_d);

	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

	// Fill n into block_size, return number of block that needed to contain all n.
	auto resolve = [](size_t n, size_t block_size)-> size_t {
		return static_cast<size_t>(std::ceil(
				static_cast<double>(n) / static_cast<double>(block_size)
			)
		);
	};

	block_size = resolve(block_size, 32)*32;
	auto grid_size = resolve(n, block_size);

	std::cout << std::format(
		"Calling vec_add_kernel with <<<{},{}>>>",
		grid_size,
		block_size
	) << std::endl;
	vec_add_kernel<<<grid_size, block_size>>>(
		a_d,
		b_d,
		c_d,
		n
	);
	hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
}

void hello_kernel_cuda() {
	int count;
	hipError_t err = hipGetDeviceCount(&count);
	printf("GPU count = %d\n", count);

	hello_kernel<<<1,1>>>();
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		printf("CUDA error: %s\n", hipGetErrorString(err));
}