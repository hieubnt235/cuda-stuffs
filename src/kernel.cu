#include "hip/hip_runtime.h"
//
// Created by hieu on 9/16/25.
//

#include <iostream>
#include <hip/hip_runtime.h>
#include<thrust/device_vector.h>

__global__ void hello_kernel() {
    printf("Hello World from GPU!\n");
}

int main() {
    // Launch kernel with 1 block, 1 thread
    hello_kernel<<<1, 1>>>();

    // Wait for GPU to finish
    hipDeviceSynchronize();

    std::cout << "Hello World from CPU!" << std::endl;
    return 0;
}
